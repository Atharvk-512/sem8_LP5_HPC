
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


__global__ void gpu_add(int *arr1, int *arr2, int *res, int size){
	int block_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(block_id < size) {
		res[block_id] = arr1[block_id] + arr2[block_id];
	}
}


void fill_array(int *arr, int size) {
	for(int i=0; i<size; i++) {
		arr[i] = rand() % 1000;
	}
}

void print_array(int *arr, int size) {
	for(int i=0; i<size; i++) cout << arr[i] << " ";
	cout<<"\n";
}


void cpu_add(int *arr1, int *arr2, int *result, int size) {
	for(int i=0;i<size;i++) {
		result[i] = arr1[i]+arr2[i];
	}
}


int main() {
	int *arr1_cpu, *arr2_cpu, *result_cpu;
	int *arr1_gpu, *arr2_gpu, *result_gpu;

	int size;

	cout << "Enter size of array: ";
	cin >> size;

	arr1_cpu = new int[size];
	arr2_cpu = new int[size];
	result_cpu = new int[size];

	fill_array(arr1_cpu,size);
	fill_array(arr2_cpu,size);

	// cout<<"Array 1: ";
	// print_array(arr1_cpu,size);

	// cout<<"Array 2: ";
	// print_array(arr2_cpu,size);

	size_t arrSize = size * sizeof(int);

	hipMalloc(&arr1_gpu, arrSize);
	hipMalloc(&arr2_gpu, arrSize);
	hipMalloc(&result_gpu, arrSize);

	hipMemcpy(arr1_gpu, arr1_cpu, arrSize, hipMemcpyHostToDevice);
	hipMemcpy(arr2_gpu, arr2_cpu, arrSize, hipMemcpyHostToDevice);


	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;


	hipEvent_t start, stop;
	float elapsed_time;


	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	gpu_add<<<blocksPerGrid, threadsPerBlock>>> (arr1_gpu, arr2_gpu, result_gpu, size);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(result_cpu, result_gpu, arrSize, hipMemcpyDeviceToHost);

	cout << "\nResult array after addition on gpu: ";
	// print_array(result_cpu,size);

	cout << "Elapsed Time: " << elapsed_time << " milliseconds\n";



	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	cpu_add(arr1_cpu, arr2_cpu, result_cpu, size);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << "\nResult array after addition on cpu: ";
	// print_array(result_cpu,size);

	cout << "Elapsed Time: " << elapsed_time << " milliseconds\n";

	hipFree(arr1_gpu);
	hipFree(arr2_gpu);
	hipFree(result_gpu);

}



// atharv@atharv-laptop-ubuntu:~/BE_sem8_HPC/Assn4_CUDA$ nvcc code.cu
// atharv@atharv-laptop-ubuntu:~/BE_sem8_HPC/Assn4_CUDA$ ./a.out 
// Enter size of array: 100000

// Result array after addition on gpu: Elapsed Time: 0.0184 milliseconds

// Result array after addition on cpu: Elapsed Time: 0.283584 milliseconds
